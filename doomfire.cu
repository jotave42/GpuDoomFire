#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

void prinrtMat(int** mat,int elem)
{
    for (int i = 0; i < elem; i++)
    {
        printf("[ ");
        for (int j = 0; j < elem; j++)
        {
            int num =mat[i][j];
        
            if(num<10)
                printf("0%d ",num);
            else
                printf("%d ",num);
        }
        printf("]\n");
    }
}

__device__ void updateFireIntensityPerPixelSimple(int** mat,int tam ,int posX, int posY)
{
    int belowPosX = posX;
    int decay = rand() % 3;
    int decayPosY = rand() % 5 + (-2); //random number between -3 and 3
    int belowPosY = posY + decayPosY;

    if(posX < tam - 1)
        belowPosX++;
    
    if(belowPosY < 0)
    {
        if(posX > 0)
        {
            posX --;
            belowPosY = tam - 1 + belowPosY;
        }
        else
            belowPosY=0;
    }
    else if(belowPosY > tam - 1)
    {
        if(posX < tam - 1)
        {
            posX ++;
            belowPosY = belowPosY - tam - 1;
        }
        else
            belowPosY = tam - 1;
    }

    int belowPixelFireIntensity = mat[belowPosX][belowPosY];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;

    mat[posX][posY]=newFireIntensity;
}

void updateFireIntensityPerPixel(int** mat,int elem ,int posX, int posY){
    
    int belowPosX = posX + 1;
    if(belowPosX == elem - 1);
        belowPosX = posX;
    
    int belowPosY = posY;
    int decay = rand() % 3;
    
    int decayPosX = rand() % 3;
    int decayPosY = rand() % 3;
    int belowPixelFireIntensity = mat[belowPosX][belowPosY];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;

    int newPosX = belowPosX + decayPosX < elem ? belowPosX + decayPosX: belowPosX;
    int newPosY = 0;
    
    if(newPosY+decayPosY< elem)
    {
        newPosY = newPosY+decayPosY;
    }
    else
    {
        if(newPosX +1 < elem - 2)
        {
            newPosX++;
            newPosY = newPosY + decayPosY;
        }
        else
            newPosY = elem - 1;
    }

    printf("mat[%d][%d]=%d\n", newPosX, newPosY, newFireIntensity);
    if(newPosX < elem - 1);
        mat[newPosX][newPosY] = newFireIntensity;

}

__global__ void calculeteFirePropagation(int** mat,int elem)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int stride_x = blockDim.x * gridDim.x;

    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int stride_y = blockDim.y * gridDim.y;

    for (int i = elem - 2; i >= 0; i--)
    {
        for (int j = 0; j < elem; j++)
        {
            updateFireIntensityPerPixelSimple<<<1, 1>>>(mat,elem ,i, j);
        }
    }
}

__global__ void creatFireSource(int** mat,int elem)
{
    int index = threadIdx.y + blockIdx.y * blockDim.y;
    int stride = blockDim.y * gridDim.y;

    int lestLine = elem - 1;
    
    for (int j = index; j < elem; j += stride)
    {
        mat[lestLine][j]= 36;
    }
}

__global__ void loadFireStruct(int** mat,int elem)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int stride_x = blockDim.x * gridDim.x;

    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int stride_y = blockDim.y * gridDim.y;

    for (int i = index_x; i < elem; i += stride_x)
    {
        for (int j = index_y; j < elem; j += stride_y)
        {
            mat[i][j]=0;
        }
    }
}

int main(int argc, char *argv[])
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
   
    size_t threadsPerBlock;
    size_t numberOfBlocks;
    
    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;
    
    int elem = 40;
    
    int size = elem * elem * sizeof(int);

    int ** fireStruct;
    hipMallocManaged (&fireStruct, size);
    
    loadFireStruct<<<numberOfBlocks, threadsPerBlock>>>(fireStruct,elem);
    hipDeviceSynchronize();

    creatFireSource<<<numberOfBlocks, threadsPerBlock>>>(fireStruct,elem);
    hipDeviceSynchronize();

    while (1)
    {
        calculeteFirePropagation<<<numberOfBlocks, threadsPerBlock>>>(fireStruct,elem);
        hipDeviceSynchronize();

        printf("=================\n");
        prinrtMat(fireStruct,elem);
        //sleep(1);
    }

    hipFree(fireStruct);

    return 0;
}