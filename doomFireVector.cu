#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <GL/glut.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

typedef struct Color 
{ 
    float red; 
    float green; 
    float blue; 
} Color_T;

Color_T ** colors;
int * fireStruct;
int line_length, col_legth, width, height, num_elem_total;
Color_T * createColor(float red, float green, float blue)
{
    int instesity =200;
    Color_T * color =(Color_T *) malloc(sizeof(Color_T));
    color->red = red / instesity;
    color->green = green / instesity;
    color->blue = blue / instesity;
    return color;
}

Color_T ** createColorVector()
{
    
    colors = (Color_T**) malloc(sizeof(Color_T)*37);
    colors[0]  = createColor(7.0f, 7.0f, 7.0f);
    colors[1]  = createColor(31.0f, 7.0f, 7.0f);
    colors[2]  = createColor(47.0f, 15.0f, 7.0f);
    colors[3]  = createColor(71.0f, 15.0f, 7.0f);
    colors[4]  = createColor(87.0f, 23.0f, 7.0f);
    colors[5]  = createColor(103.0f, 31.0f, 7.0f);
    colors[6]  = createColor(119.0f, 31.0f, 7.0f);
    colors[7]  = createColor(143.0f, 39.0f, 7.0f);
    colors[8]  = createColor(159.0f, 47.0f, 7.0f);
    colors[9]  = createColor(175.0f, 63.0f, 7.0f);
    colors[10] = createColor(191.0f, 71.0f, 7.0f);
    colors[11] = createColor(199.0f, 71.0f, 7.0f);
    colors[12] = createColor(223.0f, 79.0f, 7.0f);
    colors[13] = createColor(223.0f, 87.0f, 7.0f);
    colors[14] = createColor(223.0f, 87.0f, 7.0f);
    colors[15] = createColor(215.0f, 95.0f, 7.0f);
    colors[16] = createColor(215.0f, 95.0f, 7.0f);
    colors[17] = createColor(215.0f, 103.0f, 15.0f);
    colors[18] = createColor(207.0f, 111.0f, 15.0f);
    colors[19] = createColor(207.0f, 119.0f, 15.0f);
    colors[20] = createColor(207.0f, 127.0f, 15.0f);
    colors[21] = createColor(207.0f, 135.0f, 23.0f);
    colors[22] = createColor(199.0f, 135.0f, 23.0f);
    colors[23] = createColor(199.0f, 143.0f, 23.0f);
    colors[24] = createColor(199.0f, 151.0f, 31.0f);
    colors[25] = createColor(191.0f, 159.0f, 31.0f);
    colors[26] = createColor(191.0f, 159.0f, 31.0f);
    colors[27] = createColor(191.0f, 167.0f, 39.0f);
    colors[28] = createColor(191.0f, 167.0f, 39.0f);
    colors[29] = createColor(191.0f, 175.0f, 47.0f);
    colors[30] = createColor(183.0f, 175.0f, 47.0f);
    colors[31] = createColor(183.0f, 183.0f, 47.0f);
    colors[32] = createColor(183.0f, 183.0f, 55.0f);
    colors[33] = createColor(207.0f, 207.0f, 111.0f);
    colors[34] = createColor(223.0f, 223.0f, 159.0f);
    colors[35] = createColor(239.0f, 239.0f, 199.0f);
    colors[36] = createColor(255.0f, 255.0f, 255.0f);
    return colors;
}

void loadFireStruct(int* fire, int line_length, int col_legth)
{
    for (int line = 0; line < line_length; line++)
    {
        for (int col = 0; col < col_legth; col++)
        {
            int index = line * col_legth + col;
            if(line == line_length - 1)
                fire[index] = 36;
            else
                fire[index] = 0;
        }
    }
}

void prinrtMat(int* fire,int line_length, int col_legth){
    for (int line = 0; line < line_length; line++){
        printf("[ ");
        for (int col = 0; col < col_legth; col++){
            int index = line * col_legth + col;
            int num = fire[index];
            if(num<10)
                printf("0%d ", num);
            else
                printf("%d ", num);
        }
        printf("]\n");
    }
}


__device__ void updateFireIntensityPerPixelKernel(int* fire, int line_length, int col_legth, int currentPixelIndex)
{
    int totalOfPixels = line_length * col_legth;
    int belowPixelIndex = currentPixelIndex + col_legth;

    if(belowPixelIndex < totalOfPixels)
    {   float num_randf = 0.0f;  
        long int num;
        hiprandState state;
        hiprand_init(1234, currentPixelIndex,2, &state);

        num_randf = hiprand_uniform(&state);
        num = (long int) (num_randf*100);
        int decay =(int) (num %3);
        printf("decay => %d\n",decay);
        int decayIndex =(int) (num % 5 + (-2));
        int belowPixelFireIntensity = fire[belowPixelIndex];
        int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;
        fire[currentPixelIndex - decayIndex] = newFireIntensity;
    }
}

__global__ void calculeteFirePropagationKernel(int* fire, int line_length, int col_legth, size_t threadsPerBlock, size_t numberOfBlocks, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(n>index){
        printf("index %d\n",index);
        int stride = blockDim.x * gridDim.x;
        printf("stride %d\n",stride);
        int currentPixel = index;
        printf("Pixel %d\n",currentPixel);
        updateFireIntensityPerPixelKernel(fire, line_length, col_legth, currentPixel);
    }
   
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  
  int num_elem_line = 10;
  int num_elem_col = 10;
  int num_elem_total = num_elem_line * num_elem_col;
  
  size_t size = num_elem_total * sizeof(int);

  hipMallocManaged(&fireStruct, size);

  loadFireStruct(fireStruct, num_elem_line, num_elem_col);
  //colors = createColorVector();
  
  hipMemPrefetchAsync(fireStruct, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 32;
  numberOfBlocks = 32 * numberOfSMs;
  char ch;
  while (1)
  {
      calculeteFirePropagationKernel<<<numberOfBlocks, threadsPerBlock>>>(fireStruct, num_elem_line, num_elem_col, threadsPerBlock, numberOfBlocks,num_elem_total);

      hipDeviceSynchronize();

      prinrtMat(fireStruct, num_elem_line, num_elem_col);
      scanf("%c", &ch);
  }

  hipFree(fireStruct);
}

/*
int main(int argc, int *argv[])
{
    int num_elem_line = 40;
    int num_elem_col = 40;
    int num_elem_total = num_elem_line * num_elem_col;
    fireStruct = (int*) malloc(sizeof(int) * num_elem_total);
    loadFireStruct(fireStruct, num_elem_line, num_elem_col);
    colors = createColorVector();
    int ch;
    while (1)
    {
        calculeteFirePropagation(fireStruct, num_elem_line, num_elem_col);
        prinrtMat(fireStruct, num_elem_line, num_elem_col);
        scanf("%c", &ch);
    }    
    free(fireStruct);
} 
*/
