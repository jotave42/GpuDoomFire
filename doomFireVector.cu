#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <GL/glut.h>
#include <hip/hip_runtime.h>

typedef struct Color 
{ 
   int red; 
   int green; 
   int blue; 
} Color_T;

Color_T ** colors;

Color_T * createColor(int red, int green, int blue)
{
    Color_T * color = malloc(sizeof(Color_T));
    color->red = red;
    color->green = green;
    color->blue = blue;
    return color;
}

Color_T ** createColorVector()
{
    colors = (Color_T **) malloc(sizeof(Color_T) * 37);
    colors[0]  = createColor(7, 7, 7);
    colors[1]  = createColor(31, 7, 7);
    colors[2]  = createColor(47, 15, 7);
    colors[3]  = createColor(71, 15, 7);
    colors[4]  = createColor(87, 23, 7);
    colors[5]  = createColor(103, 31, 7);
    colors[6]  = createColor(119, 31, 7);
    colors[7]  = createColor(143, 39, 7);
    colors[8]  = createColor(159, 47, 7);
    colors[9]  = createColor(175, 63, 7);
    colors[10] = createColor(191, 71, 7);
    colors[11] = createColor(199, 71, 7);
    colors[12] = createColor(223, 79, 7);
    colors[13] = createColor(223, 87, 7);
    colors[14] = createColor(223, 87, 7);
    colors[15] = createColor(215, 95, 7);
    colors[16] = createColor(215, 95, 7);
    colors[17] = createColor(215, 103, 15);
    colors[18] = createColor(207, 111, 15);
    colors[19] = createColor(207, 119, 15);
    colors[20] = createColor(207, 127, 15);
    colors[21] = createColor(207, 135, 23);
    colors[22] = createColor(199, 135, 23);
    colors[23] = createColor(199, 143, 23);
    colors[24] = createColor(199, 151, 31);
    colors[25] = createColor(191, 159, 31);
    colors[26] = createColor(191, 159, 31);
    colors[27] = createColor(191, 167, 39);
    colors[28] = createColor(191, 167, 39);
    colors[29] = createColor(191, 175, 47);
    colors[30] = createColor(183, 175, 47);
    colors[31] = createColor(183, 183, 47);
    colors[32] = createColor(183, 183, 55);
    colors[33] = createColor(207, 207, 111);
    colors[34] = createColor(223, 223, 159);
    colors[35] = createColor(239, 239, 199);
    colors[36] = createColor(255, 255, 255);
    return colors;
}

void loadFireStruct(int* fire, int line_length, int col_legth)
{
    for (int line = 0; line < line_length; line++)
    {
        for (int col = 0; col < col_legth; col++)
        {
            int index = line * col_legth + col;
            if(line == line_length - 1)
                fire[index] = 36;
            else
                fire[index] = 0;
        }
    }
}

void prinrtMat(int* fire,int line_length, int col_legth){
    for (int line = 0; line < line_length; line++){
        printf("[ ");
        for (int col = 0; col < col_legth; col++){
            int index = line * col_legth + col;
            int num = fire[index];
            if(num<10)
                printf("0%d ", num);
            else
                printf("%d ", num);
        }
        printf("]\n");
    }
}

void updateFireIntensityPerPixel(int* fire, int line_length, int col_legth, int currentPixelIndex)
{
    int totalOfPixels = line_length * col_legth;
    int belowPixelIndex = currentPixelIndex + col_legth;

    if(belowPixelIndex >= totalOfPixels)
        return;

    int decay = rand() % 3;
    int decayIndex = rand() % 5 + (-2);
    int belowPixelFireIntensity = fire[belowPixelIndex];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;
    fire[currentPixelIndex - decayIndex] = newFireIntensity;
}

__global__ void updateFireIntensityPerPixelKernel(int* fire, int line_length, int col_legth, int currentPixelIndex)
{
    int totalOfPixels = line_length * col_legth;
    int belowPixelIndex = currentPixelIndex + col_legth;

    if(belowPixelIndex < totalOfPixels)
    {
        int decay = rand() % 3;
        int decayIndex = rand() % 5 + (-2);
        int belowPixelFireIntensity = fire[belowPixelIndex];
        int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;
        fire[currentPixelIndex - decayIndex] = newFireIntensity;
    }
}

__global__ void calculeteFirePropagationKernel(int* fire, int line_length, int col_legth, size_t threadsPerBlock, size_t numberOfBlocks)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) 
    {
        for (int j = index; j < n; j += stride)
        {
            int currentPixel = i * col_legth + j;
            updateFireIntensityPerPixelKernel<<<threadsPerBlock, numberOfBlocks>>>(fire, line_length, col_legth, currentPixel);

            hipDeviceSynchronize();
        }
    }
}

void calculeteFirePropagation(int* fire, int line_length, int col_legth)
{
    for (int line = 0; line < line_length; line++)
    {
        for (int col = 0; col < col_legth; col++)
        {
            int currentPixel = line * col_legth + col;
            updateFireIntensityPerPixel(fire, line_length, col_legth, currentPixel);
        }
    }
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  
  int num_elem_line = 40;
  int num_elem_col = 40;
  int num_elem_total = num_elem_line * num_elem_col;
  
  size_t size = num_elem_total * sizeof(int);
  int * fireStruct;

  hipMallocManaged(&fireStruct, size);

  loadFireStruct(fireStruct, num_elem_line, num_elem_col);
  //colors = createColorVector();
  
  hipMemPrefetchAsync(fireStruct, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  while (1)
  {
      calculeteFirePropagationKernel<<<numberOfBlocks, threadsPerBlock>>>(fireStruct, num_elem_line, num_elem_col, threadsPerBlock, numberOfBlocks);

      hipDeviceSynchronize();

      prinrtMat(fireStruct, num_elem_line, num_elem_col);
  }

  hipFree(fireStruct);
}

/*
int main(int argc, int *argv[])
{
    int num_elem_line = 40;
    int num_elem_col = 40;
    int num_elem_total = num_elem_line * num_elem_col;
    fireStruct = (int*) malloc(sizeof(int) * num_elem_total);
    loadFireStruct(fireStruct, num_elem_line, num_elem_col);
    colors = createColorVector();
    int ch;
    while (1)
    {
        calculeteFirePropagation(fireStruct, num_elem_line, num_elem_col);
        prinrtMat(fireStruct, num_elem_line, num_elem_col);
        scanf("%c", &ch);
    }    
    free(fireStruct);
} 
*/
