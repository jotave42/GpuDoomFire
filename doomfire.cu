#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

void prinrtMat(int** mat,int elem)
{
    for (int i = 0; i < elem; i++)
    {
        printf("[ ");
        for (int j = 0; j < elem; j++)
        {
            int num =mat[i][j];
        
            if(num<10)
                printf("0%d ",num);
            else
                printf("%d ",num);
        }
        printf("]\n");
    }
}

void updateFireIntensityPerPixelSimple(int** mat,int tam ,int posX, int posY)
{
    int belowPosX = posX;
    int decay = rand() % 3;
    int decayPosY = rand() % 5 + (-2); //random number between -3 and 3
    int belowPosY = posY + decayPosY;

    if(posX < tam - 1)
        belowPosX++;
    
    if(belowPosY < 0)
    {
        if(posX > 0)
        {
            posX --;
            belowPosY = tam - 1 + belowPosY;
        }
        else
            belowPosY=0;
    }
    else if(belowPosY > tam - 1)
    {
        if(posX < tam - 1)
        {
            posX ++;
            belowPosY = belowPosY - tam - 1;
        }
        else
            belowPosY = tam - 1;
    }

    int belowPixelFireIntensity = mat[belowPosX][belowPosY];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;

    mat[posX][posY]=newFireIntensity;
}

void updateFireIntensityPerPixel(int** mat,int elem ,int posX, int posY){
    
    int belowPosX = posX + 1;
    if(belowPosX == elem - 1);
        belowPosX = posX;
    
    int belowPosY = posY;
    int decay = rand() % 3;
    
    int decayPosX = rand() % 3;
    int decayPosY = rand() % 3;
    int belowPixelFireIntensity = mat[belowPosX][belowPosY];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;

    int newPosX = belowPosX + decayPosX < elem ? belowPosX + decayPosX: belowPosX;
    int newPosY = 0;
    
    if(newPosY+decayPosY< elem)
    {
        newPosY = newPosY+decayPosY;
    }
    else
    {
        if(newPosX +1 < elem - 2)
        {
            newPosX++;
            newPosY = newPosY + decayPosY;
        }
        else
            newPosY = elem - 1;
    }

    printf("mat[%d][%d]=%d\n", newPosX, newPosY, newFireIntensity);
    if(newPosX < elem - 1);
        mat[newPosX][newPosY] = newFireIntensity;

}

__global__ void calculeteFirePropagation(int ** mat,int elem)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = elem - 2; i >= 0; i--)
    {
        for (int j = 0; j < elem; j++)
        {
            updateFireIntensityPerPixelSimple(mat,elem ,i, j);
        }
    }
}

__global__ void creatFireSource(int** mat,int elem)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int lestLine = elem - 1;
    
    for (int j = index; j < elem; j += stride)
    {
        mat[lestLine][j]= 36;
    }
}

__global__ void loadFireStruct(int** mat,int elem)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < elem; i += stride)
    {
        for (int j = index; j < elem; j += stride)
        {
            mat[i][j]=0;
        }
    }
}

int main(int argc, char *argv[])
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
   
    size_t threadsPerBlock;
    size_t numberOfBlocks;
    
    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;
    
    int elem = 40;
    
    int size = elem*elem*sizeof(int);
    int * fireStruct;
    hipMallocManaged (&fireStruct, size);
    

    loadFireStruct(fireStruct,elem);
    creatFireSource(fireStruct,elem);
    while (1)
    {
        calculeteFirePropagation(fireStruct,elem);
        printf("=================\n");
        prinrtMat(fireStruct,elem);
        sleep(1);
    }
    hipFree(fireStruct);
    //REMEMBER CUDAFREE hipFree();

    return 0;
}