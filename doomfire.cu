#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>


inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    //assert(result == cudaSuccess);
  }
  return result;
}

void prinrtMat(int** mat,int elem)
{
    for (int i = 0; i < elem; i++)
    {
        printf("[ ");
        for (int j = 0; j < elem; j++)
        {
            int num =mat[i][j];
        
            if(num<10)
                printf("0%d ",num);
            else
                printf("%d ",num);
        }
        printf("]\n");
    }
}

__device__ void updateFireIntensityPerPixelSimple(int** mat,int tam ,int posX, int posY,int index)
{
    int belowPosX = posX;
    //int decay = rand() % 3;
    //int decayPosY = rand() % 5 + (-2); //random number between -3 and 3
    //int belowPosY = posY + decayPosY;
    
    hiprandState state;

    hiprand_init(1234, index, 0, &state);
    float res = hiprand_uniform(&state);
    printf("res = %f\n",res);
    return;

  /*  if(posX < tam - 1)
        belowPosX++;
    
    if(belowPosY < 0)
    {
        if(posX > 0)
        {
            posX --;
            belowPosY = tam - 1 + belowPosY;
        }
        else
            belowPosY=0;
    }
    else if(belowPosY > tam - 1)
    {
        if(posX < tam - 1)
        {
            posX ++;
            belowPosY = belowPosY - tam - 1;
        }
        else
            belowPosY = tam - 1;
    }

    int belowPixelFireIntensity = mat[belowPosX][belowPosY];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;

    mat[posX][posY]=newFireIntensity;*/
}

void updateFireIntensityPerPixel(int** mat,int elem ,int posX, int posY){
    
    int belowPosX = posX + 1;
    if(belowPosX == elem - 1);
        belowPosX = posX;
    
    int belowPosY = posY;
    int decay = rand() % 3;
    
    int decayPosX = rand() % 3;
    int decayPosY = rand() % 3;
    int belowPixelFireIntensity = mat[belowPosX][belowPosY];
    int newFireIntensity = belowPixelFireIntensity - decay >= 0 ? belowPixelFireIntensity - decay: 0;

    int newPosX = belowPosX + decayPosX < elem ? belowPosX + decayPosX: belowPosX;
    int newPosY = 0;
    
    if(newPosY+decayPosY< elem)
    {
        newPosY = newPosY+decayPosY;
    }
    else
    {
        if(newPosX +1 < elem - 2)
        {
            newPosX++;
            newPosY = newPosY + decayPosY;
        }
        else
            newPosY = elem - 1;
    }

    printf("mat[%d][%d]=%d\n", newPosX, newPosY, newFireIntensity);
    if(newPosX < elem - 1);
        mat[newPosX][newPosY] = newFireIntensity;

}

__global__ void calculeteFirePropagation(int** mat,int elem)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int stride_x = blockDim.x * gridDim.x;

    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int stride_y = blockDim.y * gridDim.y;

    for (int i = elem - 2; i >= 0; i--)
    {
        for (int j = 0; j < elem; j++)
        {
            updateFireIntensityPerPixelSimple(mat,elem ,i, j, index_x);
        }
    }
}

__global__ void creatFireSource(int** mat,int elem)
{
    int index = threadIdx.y + blockIdx.y * blockDim.y;
    int stride = blockDim.y * gridDim.y;

    int lestLine = elem - 1;
    
    for (int j = index; j < elem; j += stride)
    {
        mat[lestLine][j]= 36;
    }
}

__global__ void loadFireStruct(int** mat,int elem)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int stride_x = blockDim.x * gridDim.x;

    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int stride_y = blockDim.y * gridDim.y;

    for (int i = index_x; i < elem; i += stride_x)
    {
        for (int j = index_y; j < elem; j += stride_y)
        {
            mat[i][j]=0;
        }
    }
}

int main(int argc, char *argv[])
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
   
    size_t threadsPerBlock;
    size_t numberOfBlocks;
    
    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;
    
    int elem = 40;
    
    int size = elem * elem * sizeof(int);

    int ** fireStruct;
    printf("fazendo malloc ...\n");
    checkCuda(hipMallocManaged((void**)&fireStruct, size));
    
    printf("fazendo chaamando loadFireStruct ...\n");
    loadFireStruct<<<numberOfBlocks, threadsPerBlock>>>(fireStruct,elem);
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    printf("fazendo chamando creatFireSource ...\n");
    creatFireSource<<<numberOfBlocks, threadsPerBlock>>>(fireStruct,elem);
    checkCuda(hipGetLastError() );
    checkCuda(hipDeviceSynchronize());

    while (1)
    {
        printf("fazendo chamando calculeteFirePropagation ...\n");
        calculeteFirePropagation<<<numberOfBlocks, threadsPerBlock>>>(fireStruct,elem);
        checkCuda(hipGetLastError() );
        checkCuda(hipDeviceSynchronize());

        printf("=================\n");
        prinrtMat(fireStruct,elem);
        //sleep(1);
    }
    printf("fazendo chamando hipFree ...\n");
    checkCuda(hipFree(fireStruct));

    return 0;
}